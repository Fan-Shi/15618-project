#include <algorithm>
#include <vector>
#include <random>
#include <functional>
#include <iostream>
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <mma.h>
#include <hip/hip_fp16.h>

#define TILE_WIDTH 16
#define WARP_SIZE 32

#define GPU_ERROR(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
   if (code != hipSuccess) {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

using namespace nvcuda;

__global__ void gemmBasicKernelFp16(const half *A, const half *B, half *C, int M, int N, int K) {
    int colIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int rowIdx = blockIdx.y * blockDim.y + threadIdx.y;

    if (rowIdx < M && colIdx < N) {
        half val = __float2half(0.0f);
        for(int kdx = 0; kdx < K; kdx++) {
            val = __hfma(A[rowIdx * K + kdx], B[kdx * N + colIdx], val); // Notice matrix B is transposed
        }
        C[rowIdx * N + colIdx] = val;
    }
}

__global__ void gemmTensorCoreKernelFp16(const half *A, const half *B, half *C, int M, int N, int K) {
    // Each block contains 4 warps, with 2 x 2 latout. 
    // Each Warp contain 32 threads, and will be responsible for compute 16 x 16 output tile using tensor core
    // Then, each block will output (2 x 16) x (2 x 16) = 32 x 32 output tiles 
    int tileAIndex = (blockIdx.x * blockDim.x + threadIdx.x) / WARP_SIZE;
    int tileBIndex = (blockIdx.y * blockDim.y + threadIdx.y);

    // Matrix A is stored in row major layout of shape M * K. 
    // Matrix B is stored in col major layout of shale N * K to simplify programming  
    // Each output tile has shape 16 * 16. initialzie to 0
    wmma::fragment<wmma::matrix_a, TILE_WIDTH, TILE_WIDTH, TILE_WIDTH, half, wmma::row_major> aFrag;
    wmma::fragment<wmma::matrix_b, TILE_WIDTH, TILE_WIDTH, TILE_WIDTH, half, wmma::col_major> bFrag;
    wmma::fragment<wmma::accumulator, TILE_WIDTH, TILE_WIDTH, TILE_WIDTH, half> valFrag;
    wmma::fill_fragment(valFrag, __float2half(0.0f));

    int rowAStartIndex = tileAIndex * TILE_WIDTH;
    int rowBStartIndex = tileBIndex * TILE_WIDTH;

    for(int kdx = 0; kdx < K; kdx += TILE_WIDTH) {
        // Assume the matrix has perfect dimension. We don't check bound here
        wmma::load_matrix_sync(aFrag, A + rowAStartIndex * K + kdx, K /*leading index*/); 
        wmma::load_matrix_sync(aFrag, B + rowBStartIndex * K + kdx, K /*leading index*/); 

        // Multiplication perfromed on ATile * BTile. Notice B is loaded as col major layout
        wmma::mma_sync(valFrag, aFrag, bFrag, valFrag);
    }

    // The upper left position of output matrix is [rowAStartIndex, rowBStartIndex] 
    wmma::store_matrix_sync(C + rowBStartIndex + rowAStartIndex * N, valFrag, N, wmma::mem_row_major);
}

void runExperimentFp16(int m, int n, int k) {
    std::vector<half> aVec(m * k);
    std::vector<half> bVec(n * k);
    std::vector<half> cVec(m * n, __float2half(0.0f));

    // initialize matrix with random values
    std::uniform_real_distribution<float> distribution(0.0f, 5.0f);
    std::mt19937 engine;
    std::generate(aVec.begin(), aVec.end(), [&distribution, &engine](){
        return __float2half(distribution(engine));
    });
    std::generate(bVec.begin(), bVec.end(), [&distribution, &engine](){
        return __float2half(distribution(engine));
    });

    // malloc device memory and initialize them
    half *deviceA, *deviceB, *deviceC;
    GPU_ERROR(hipMalloc(&deviceA, m * k * sizeof(half)));
    GPU_ERROR(hipMalloc(&deviceB, k * n * sizeof(half)));
    GPU_ERROR(hipMalloc(&deviceC, m * n * sizeof(half)));

    GPU_ERROR(hipMemcpy(deviceA, aVec.data(), m * k * sizeof(half), hipMemcpyHostToDevice));
    GPU_ERROR(hipMemcpy(deviceB, bVec.data(), k * n * sizeof(half), hipMemcpyHostToDevice));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    dim3 gridDimBasic(m / TILE_WIDTH, n / TILE_WIDTH, 1);
    dim3 blockDimBasic(TILE_WIDTH, TILE_WIDTH, 1);
    hipEventRecord(start, 0);
    gemmBasicKernelFp16<<<gridDimBasic, blockDimBasic>>>(deviceA, deviceB, deviceC, m, n, k);
    hipEventRecord(stop, 0);

    hipEventSynchronize(stop);
    float basicGemmFp16Millis = 0.0f;
    hipEventElapsedTime(&basicGemmFp16Millis, start, stop);
    std::cout << "case " << k <<" BASIC GEMM gpu take " << basicGemmFp16Millis << " to complete." << std::endl;

    hipError_t error = hipGetLastError();
    if(error!=hipSuccess) {
        fprintf(stderr,"ERROR Basic : %s\n", hipGetErrorString(error) );
        exit(-1);
    }

    dim3 gridDim(m / (2 * TILE_WIDTH), n / (2 * TILE_WIDTH), 1);
    dim3 blockDim(2 * WARP_SIZE, 2, 1);


    hipEventRecord(start, 0);
    gemmTensorCoreKernelFp16<<<gridDim, blockDim>>>(deviceA, deviceB, deviceC, m, n, k);
    hipEventRecord(stop, 0);

    hipEventSynchronize(stop);
    float tensorCoreGemmFp16Millis = 0.0f;
    hipEventElapsedTime(&tensorCoreGemmFp16Millis, start, stop);
    std::cout << "case " << k <<" Tensor Core GEMM gpu take " << tensorCoreGemmFp16Millis << " to complete." << std::endl;

    error = hipGetLastError();
    if(error!=hipSuccess) {
        fprintf(stderr,"ERROR WMMA : %s\n", hipGetErrorString(error) );
        exit(-1);
    }
}

int main(int argc, char** argv) {
    runExperimentFp16(64, 64, 64);
    runExperimentFp16(128, 128, 128);
    runExperimentFp16(256, 256, 256);
    runExperimentFp16(512, 512, 512);
    runExperimentFp16(1024, 1024, 1024);
}

